#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <cmath> // Include cmath for fabs
using namespace std;
using namespace std::chrono;
#define N 10000000
#define M 30

#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __LINE__ << endl; \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void matrixMul(float *A, float *B, float *C, int m) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < m && col < m) {
        float sum = 0;
        for (int k = 0; k < m; ++k) {
            sum += A[row * m + k] * B[k * m + col];
        }
        C[row * m + col] = sum;
    }
}

int main() {
    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
    h_a = new float[N];
    h_b = new float[N];
    h_c = new float[N];
    vector<float> cpu_c_vector(N);

    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    CHECK_CUDA_ERROR(hipMalloc(&d_a, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, N * sizeof(float)));

    auto start_cpu = high_resolution_clock::now();
    for (int i = 0; i < N; i++) {
        cpu_c_vector[i] = h_a[i] + h_b[i];
    }
    auto end_cpu = high_resolution_clock::now();
    double cpu_time = duration<double>(end_cpu - start_cpu).count();

    hipEvent_t start, stop;
    float gpu_time;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));

    hipEventRecord(start);
    vectorAdd<<< (N + 255) / 256, 256 >>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost));
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    cout << "Vector Addition CPU time: " << cpu_time << " sec\n";
    cout << "Vector Addition GPU time: " << gpu_time / 1000.0 << " sec\n";
    cout << "Speedup: " << cpu_time / (gpu_time / 1000.0) << "\n";

    bool vector_match = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c[i] - cpu_c_vector[i]) > 1e-5) {
            vector_match = false;
            break;
        }
    }
    cout << "Vector Addition Results: " << (vector_match ? "Matched" : "Not Matched") << "\n\n";

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[M * M];
    h_B = new float[M * M];
    h_C = new float[M * M];
    vector<float> cpu_C_vector(M * M);

    for (int i = 0; i < M * M; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    CHECK_CUDA_ERROR(hipMalloc(&d_A, M * M * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, M * M * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, M * M * sizeof(float)));

    start_cpu = high_resolution_clock::now();
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            float sum = 0;
            for (int k = 0; k < M; k++) {
                sum += h_A[i * M + k] * h_B[k * M + j];
            }
            cpu_C_vector[i * M + j] = sum;
        }
    }
    end_cpu = high_resolution_clock::now();
    cpu_time = duration<double>(end_cpu - start_cpu).count();

    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * M * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, M * M * sizeof(float), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEventRecord(start);
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M);
    hipEventRecord(stop);

    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * M * sizeof(float), hipMemcpyDeviceToHost));
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    cout << "Matrix Multiplication CPU time: " << cpu_time << " sec\n";
    cout << "Matrix Multiplication GPU time: " << gpu_time / 1000.0 << " sec\n";
    cout << "Speedup: " << cpu_time / (gpu_time / 1000.0) << "\n";

    bool matrix_match = true;
    for (int i = 0; i < M * M; i++) {
        if (fabs(h_C[i] - cpu_C_vector[i]) > 1e-5) {
            matrix_match = false;
            break;
        }
    }
    cout << "Matrix Multiplication Results: " << (matrix_match ? "Matched" : "Not Matched") << "\n";

    // Cleanup
    delete[] h_a, h_b, h_c, h_A, h_B, h_C;

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

// delete[] h_a;
// delete[] h_b;
// delete[] h_c;
// delete[] h_A;
// delete[] h_B;
// delete[] h_C;
